/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "bitmask/bitmask.h"
#include "cudf_util/detail.h"
#include "util/cuda_helper.h"

#include <cudf/reduction.hpp>
#include <cudf/column/column_view.hpp>

#include <thrust/transform.h>

#include <rmm/exec_policy.hpp>

namespace legate {
namespace pandas {

void Bitmask::set_all_valid(hipStream_t stream)
{
  hipMemsetAsync(bitmask, 0x01, num_elements, stream);
}

void Bitmask::clear(hipStream_t stream) { hipMemsetAsync(bitmask, 0x00, num_elements, stream); }

size_t Bitmask::count_unset_bits(hipStream_t stream) const
{
  cudf::column_view boolmask{
    cudf::data_type{cudf::type_id::UINT8}, static_cast<cudf::size_type>(num_elements), bitmask};
  auto type_id                        = cudf::data_type{cudf::type_to_id<int32_t>()};
  rmm::mr::device_memory_resource *mr = rmm::mr::get_current_device_resource();
  auto out = cudf::detail::reduce(boolmask, cudf::make_sum_aggregation(), type_id, stream, mr);
  auto null_count = static_cast<cudf::scalar_type_t<int32_t> *>(out.get())->value(stream);
  assert(num_elements >= null_count);
  return num_elements - null_count;
}

void Bitmask::copy(const Bitmask &target, hipStream_t stream) const
{
  hipMemcpyAsync(target.bitmask, bitmask, num_elements, hipMemcpyDeviceToDevice, stream);
}

void intersect_bitmasks(Bitmask &out, const Bitmask &in1, const Bitmask &in2, hipStream_t stream)
{
  auto start = thrust::make_counting_iterator<int64_t>(0);
  auto stop  = thrust::make_counting_iterator<int64_t>(static_cast<int64_t>(out.num_elements));

  thrust::for_each(rmm::exec_policy(stream), start, stop, [out, in1, in2] __device__(auto idx) {
    out.set(idx, in1.get(idx) && in2.get(idx));
  });
}

}  // namespace pandas
}  // namespace legate
