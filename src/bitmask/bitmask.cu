#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include "bitmask/bitmask.h"
#include "cudf_util/detail.h"
#include "util/cuda_helper.h"

#include <cudf/reduction.hpp>
#include <cudf/column/column_view.hpp>

namespace legate {
namespace pandas {

void Bitmask::set_all_valid(hipStream_t stream)
{
  hipMemsetAsync(bitmask, 0x01, num_elements, stream);
}

void Bitmask::clear(hipStream_t stream) { hipMemsetAsync(bitmask, 0x00, num_elements, stream); }

size_t Bitmask::count_unset_bits(hipStream_t stream) const
{
  cudf::column_view boolmask{
    cudf::data_type{cudf::type_id::UINT8}, static_cast<cudf::size_type>(num_elements), bitmask};
  auto type_id                        = cudf::data_type{cudf::type_to_id<int32_t>()};
  rmm::mr::device_memory_resource *mr = rmm::mr::get_current_device_resource();
  auto out = cudf::detail::reduce(boolmask, cudf::make_sum_aggregation(), type_id, stream, mr);
  auto null_count = static_cast<cudf::scalar_type_t<int32_t> *>(out.get())->value(stream);
  assert(num_elements >= null_count);
  return num_elements - null_count;
}

void Bitmask::copy(const Bitmask &target, hipStream_t stream) const
{
  hipMemcpyAsync(target.bitmask, bitmask, num_elements, hipMemcpyDeviceToDevice, stream);
}

static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  intersect_inplace(Bitmask out, Bitmask in)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= out.num_elements) return;
  out.set(idx, out.get(idx) && in.get(idx));
}

static __global__ void __launch_bounds__(THREADS_PER_BLOCK, MIN_CTAS_PER_SM)
  intersect(Bitmask out, Bitmask in1, Bitmask in2)
{
  const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= out.num_elements) return;
  out.set(idx, in1.get(idx) && in2.get(idx));
}

void intersect_bitmasks(Bitmask &out, const Bitmask &in1, const Bitmask &in2, hipStream_t stream)
{
  const size_t blocks = (out.num_elements + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  if (in1.bitmask == out.bitmask)
    intersect_inplace<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(out, in2);
  else
    intersect<<<blocks, THREADS_PER_BLOCK, 0, stream>>>(out, in1, in2);
}

}  // namespace pandas
}  // namespace legate
